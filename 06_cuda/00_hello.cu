
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void print(void) {
  printf("Hello GPU\n");
}

int main() {
  printf("Hello CPU\n");
  print<<<2,2>>>();
  hipDeviceSynchronize();
}
